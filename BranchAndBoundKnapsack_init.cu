#include "BranchAndBoundKnapsack_dev.cuh"

#include "hostCode/AppDriver.cuh"

BranchAndBoundKnapsack::BranchAndBoundKnapsack(hipStream_t stream, int deviceId)
 :
   SinkNode(&allParams.n__MTR_SINK_12722640[0])
{
   if (deviceId == -1) hipGetDevice(&deviceId);
   driver = new Mercator::AppDriver<Params, BranchAndBoundKnapsack_dev>(stream, deviceId);
}
