#include "BranchAndBoundKnapsack_dev.cuh"

#include "hostCode/AppDriver.cuh"

BranchAndBoundKnapsack::BranchAndBoundKnapsack(hipStream_t stream, int deviceId)
 :
   SinkNode(&allParams.n__MTR_SINK_14410064[0])
{
   if (deviceId == -1) hipGetDevice(&deviceId);
   driver = new Mercator::AppDriver<Params, BranchAndBoundKnapsack_dev>(stream, deviceId);
}
