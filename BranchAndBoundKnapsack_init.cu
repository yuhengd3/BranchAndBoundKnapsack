#include "BranchAndBoundKnapsack_dev.cuh"

#include "hostCode/AppDriver.cuh"

BranchAndBoundKnapsack::BranchAndBoundKnapsack(hipStream_t stream, int deviceId)
 :
   SinkNode(&allParams.n__MTR_SINK_12877600[0])
{
   if (deviceId == -1) hipGetDevice(&deviceId);
   driver = new Mercator::AppDriver<Params, BranchAndBoundKnapsack_dev>(stream, deviceId);
}
