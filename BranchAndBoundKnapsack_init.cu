#include "BranchAndBoundKnapsack_dev.cuh"

#include "hostCode/AppDriver.cuh"

BranchAndBoundKnapsack::BranchAndBoundKnapsack(hipStream_t stream, int deviceId)
 :
   SinkNode(&allParams.n__MTR_SINK_17682672[0])
{
   if (deviceId == -1) hipGetDevice(&deviceId);
   driver = new Mercator::AppDriver<Params, BranchAndBoundKnapsack_dev>(stream, deviceId);
}
