#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cmath>
#include <stack>
#include <vector>
#include <algorithm>
#include <iterator>

#include "SubProblem.cuh"
#include "BranchAndBoundKnapsack.cuh"

//#define MAX_ITEMS 100000
//#define MAX_CAPACITY 10000
#define HOST_MAX_ITEM 16
unsigned int OUTPUTS_MULTIPLIER = 256; // (1 << 9);
unsigned int MAX_INPUT_ = 200000;
unsigned int HOST_MAX_LEVEL = 16;
double globalLowerBound = 0;

double cpuGlobalLowerBound = 0;

unsigned int srand_seed = 0;
unsigned int MAX_ITEMS = 100000;
double MAX_CAPACITY = 10000;

unsigned counter = 0;

double calculateUpperBound(unsigned int currentItem, double currentWeight, double currentProfit, unsigned int* weights, unsigned int* profits);

void branchCPU(SubProblem s, unsigned int* weights, unsigned int* profits) {
	//Sub-problem is overweight, terminate
	if(s.currentTotalWeight > MAX_CAPACITY) {
		//cout << "Over Capacity  . . ." << endl;
		return;
	}

	counter ++;

	//Sub-problem does not do better than current globalUpperBound
	if(s.upperBound < cpuGlobalLowerBound) {
		//cout << "Upper bound lower than current best . . ." << endl;
		return;
	}

	//If we've reached a leaf node . . .
	if(s.currentItem == MAX_ITEMS) {
		double finalBranchCost = s.currentTotalProfit;
		if(finalBranchCost > cpuGlobalLowerBound) {
			cpuGlobalLowerBound = finalBranchCost;
			// globalBestSubProblem = s;

			//cout << "New Best Branch Profit: " << finalBranchCost << endl;
		}
		return;
	}

	//NOTE: WE DON'T STORE WHICH BRANCH THIS IS, SO WE JUST GO AHEAD AND RECALCULATE
	//THE UPPER BOUND EVERY TIME.  IF WE TRACKED IT, THEN WE COULD RECALCUATE FOR EVERY
	//RIGHT BRANCH ONLY INSTEAD. . .
	//if(s.storedItems[s.currentItem] == false) {
		s.upperBound = calculateUpperBound(s.currentItem, s.currentTotalWeight, s.currentTotalProfit, weights, profits);
	//}

	if(s.upperBound > cpuGlobalLowerBound) {
		SubProblem nextLeft = s;	//Include next item
		SubProblem nextRight = s;	//Exclude next item

		nextLeft.currentItem += 1;
		nextRight.currentItem += 1;

		nextLeft.currentTotalProfit += profits[nextLeft.currentItem];
		nextLeft.currentTotalWeight += weights[nextLeft.currentItem];

		//cout << endl << "Left Branch. . . " << endl;
		branchCPU(nextLeft, weights, profits);
		//cout << endl << "Right Branch. . . " << endl;
		branchCPU(nextRight, weights, profits);
	}
}

double calculateInitialLowerBound(unsigned int * weights, unsigned int * profits) {
	double currProfit = 0;
	double currWeight = 0;

	for (unsigned i = 1; i != MAX_ITEMS; i++) {
		if (currWeight + weights[i] <= MAX_CAPACITY) {
			currWeight += weights[i];
			currProfit += profits[i];
		}
	}
	return currProfit;
}

double calculateUpperBound(unsigned int currentItem, double currentWeight, double currentProfit, unsigned int* weights, unsigned int* profits) {
	double upperBoundProfit = currentProfit;
	double upperBoundWeight = currentWeight;

	unsigned int i = currentItem + 1;
	while (i < MAX_ITEMS && upperBoundWeight + weights[i] < MAX_CAPACITY) {
		upperBoundProfit += profits[i];
		upperBoundWeight += weights[i];
		++i;
	}

	if (upperBoundWeight < MAX_CAPACITY && i < MAX_ITEMS) {
		double partialCapacity = MAX_CAPACITY - upperBoundWeight;
		double percentage = double(partialCapacity) / double(weights[i]);
		upperBoundProfit += percentage * profits[i];
	}

	return upperBoundProfit;
}

void randomItems(unsigned int * weights, unsigned int * profits) {
	unsigned int baseWeights[MAX_ITEMS];
	unsigned int baseProfits[MAX_ITEMS];

	double profitPerWeight[MAX_ITEMS];

	unsigned int minWeight = 1;
	unsigned int maxWeight = 1000;
	// unsigned int minProfit = 1;
	// unsigned int maxProfit = 100;

	srand(srand_seed);

	baseWeights[0] = 0;
	baseProfits[0] = 0;
	profitPerWeight[0] = 0.0;

	for (unsigned int i = 1; i < MAX_ITEMS; ++i) {
		baseWeights[i] = rand() % (maxWeight - minWeight) + minWeight;
		// baseProfits[i] = rand() % (maxProfit - minProfit) + minProfit;
		baseProfits[i] = baseWeights[i] + 50;

		profitPerWeight[i] = double(baseProfits[i]) / double(baseWeights[i]);
	}

	// sort
	unsigned int j = 1;
	while (true) {
		double highestProfitPerWeight = 0.0;
		unsigned int index = 0;
		for (unsigned int i = 1; i < MAX_ITEMS; ++i) {
			if (highestProfitPerWeight < profitPerWeight[i]) {
				highestProfitPerWeight = profitPerWeight[i];
				index = i;
			}
		}


		weights[j] = baseWeights[index];
		profits[j] = baseProfits[index];
		//
		// baseWeights[index] = baseWeights[j];
		// baseProfits[index] = baseProfits[j];

		// profitPerWeight[index] = baseProfits[index] / baseWeights[index];
		profitPerWeight[index] = 0;
		//
		++j;

		if (j == MAX_ITEMS) {
			break;
		}
	}

	weights[0] = 0;
	profits[0] = 0;
}

int findFirstRemaining(std::vector<SubProblem> repo[]) {
	int index = MAX_ITEMS / 8;
	index -= 1;
	while (index >= 0) {
		if (!repo[index].empty()) {
			break;
		}
		index -= 1;
	}
	return index;
}

void branch(SubProblem s, unsigned int* weights, unsigned int* profits, std::vector<SubProblem> repo[]) {
	//printSubProblem(s, weights, profits);

	//Sub-problem is overweight, terminate
	if(s.currentTotalWeight > MAX_CAPACITY) {
		//cout << "Over Capacity  . . ." << endl;
		return;
	}

	//Sub-problem does not do better than current globalLowerBound
	if(s.upperBound < globalLowerBound) {
		//cout << "Upper bound lower than current best . . ." << endl;
		return;
	}

	//NOTE: WE DON'T STORE WHICH BRANCH THIS IS, SO WE JUST GO AHEAD AND RECALCULATE
	//THE UPPER BOUND EVERY TIME.  IF WE TRACKED IT, THEN WE COULD RECALCUATE FOR EVERY
	//RIGHT BRANCH ONLY INSTEAD. . .
	//if(s.storedItems[s.currentItem] == false) {
		s.upperBound = calculateUpperBound(s.currentItem, s.currentTotalWeight, s.currentTotalProfit, weights, profits);
	//}

	if(s.upperBound > globalLowerBound) {
		SubProblem nextLeft = s;	//Include next item
		SubProblem nextRight = s;	//Exclude next item

		nextLeft.currentItem += 1;
		nextRight.currentItem += 1;

		nextLeft.currentTotalProfit += profits[nextLeft.currentItem];
		nextLeft.currentTotalWeight += weights[nextLeft.currentItem];

		nextLeft.upperBound = calculateUpperBound(nextLeft.currentItem, nextLeft.currentTotalWeight, nextLeft.currentTotalProfit, weights, profits);
		nextRight.upperBound = calculateUpperBound(nextRight.currentItem, nextRight.currentTotalWeight, nextRight.currentTotalProfit, weights, profits);

	 	if (nextLeft.upperBound > globalLowerBound) {	
			if (nextLeft.currentItem < HOST_MAX_LEVEL) {
				branch(nextLeft, weights, profits, repo);
			} else {
				repo[HOST_MAX_LEVEL / 8].push_back(nextLeft);
			}
		}	
		if (nextRight.upperBound > globalLowerBound) {
		       	if (nextRight.currentItem < HOST_MAX_LEVEL) {
		       		branch(nextRight, weights, profits, repo);
		 	} else {
				repo[HOST_MAX_LEVEL / 8].push_back(nextRight);
			}	
		}
	}
}


int main(int argc, char * argv[]) {
	if (argc != 3) {
		printf("usage: ./Knapsack srand_seed max_items\n");
		return -1;
	}

	srand_seed = atoi(argv[1]);
	MAX_ITEMS = atoi(argv[2]);
	// MAX_CAPACITY = atoi(argv[3]);

	unsigned int weights[MAX_ITEMS];
	unsigned int profits[MAX_ITEMS];

	randomItems(weights, profits);
	MAX_CAPACITY = 0;
	for (unsigned i = 1; i != MAX_ITEMS; i++) {
		MAX_CAPACITY += weights[i];
		// std::cout << (double) profits[i] / weights[i] << std::endl;
	}
	MAX_CAPACITY /= 2;

	std::cout << "finished randomItems" << std::endl;

	// CPU version
	SubProblem s;
	s.currentItem = 0;
	s.currentTotalProfit = 0;
	s.currentTotalWeight = 0;

	s.upperBound = calculateUpperBound(s.currentItem, s.currentTotalWeight, s.currentTotalProfit, weights, profits);

	branchCPU(s, weights, profits);

	std::cout << "finished cpu version: " << cpuGlobalLowerBound << std::endl;	
	std::cout << "counter: " << counter << std::endl;

	globalLowerBound = calculateInitialLowerBound(weights, profits);
	std::cout << "initial global lower bound: " << globalLowerBound << std::endl;


	std::vector<SubProblem> repo[MAX_ITEMS / 8] = {std::vector<SubProblem>()};	
	SubProblem input;
	input.currentItem = 0;
	input.currentTotalProfit = 0;
	input.currentTotalWeight = 0;
	input.upperBound = calculateUpperBound(0, 0, 0, weights, profits);

	branch(input, weights, profits, repo);
       
	unsigned int input_size;
	unsigned int output_size;
	SubProblem * input_ptr = NULL;
	SubProblem * output_ptr = NULL;
	std::vector<SubProblem> leafSubProblems;
	unsigned int num_blocks = 0;
	double * block_bounds = NULL;

	// copy weights and profits to gpu memory
	/*
	unsigned * d_weights, * d_profits;
	hipError_t cudaStatus;
	cudaStatus = hipMalloc((void**) &d_weights, MAX_ITEMS * sizeof(unsigned));
       	if (cudaStatus != hipSuccess) {
		std::cout << "hipMalloc error" << std::endl;
		hipFree(d_weights);
		return -1;
	}

	cudaStatus = hipMalloc((void**) &d_profits, MAX_ITEMS * sizeof(unsigned));
	if (cudaStatus != hipSuccess) {
		std::cout << "hipMalloc error" << std::endl;
		hipFree((void*)d_profits);
		return -1;
	}
	*/
	int index = -1;

	while ((index = findFirstRemaining(repo)) > -1) {
		std::cout << "index: " << index << std::endl;
		std::vector<SubProblem> & nextVec = repo[index];
		if (nextVec.size() <= MAX_INPUT_) {
			input_size = nextVec.size();
			input_ptr = new SubProblem[input_size];
			std::copy(nextVec.begin(), nextVec.end(), input_ptr);
			nextVec.clear();
		} else {
			input_size = MAX_INPUT_;
			input_ptr = new SubProblem[input_size];
			std::copy(nextVec.begin(), nextVec.begin()+MAX_INPUT_, input_ptr);
			// delete from the start of the vector
			nextVec.erase(nextVec.begin(), nextVec.begin()+MAX_INPUT_);
		}

		output_size = input_size * OUTPUTS_MULTIPLIER;
		// std::cout << "output size " << output_size << std::endl;
		output_ptr = new SubProblem[output_size];

		Mercator::Buffer<SubProblem> inBuffer(input_size);
		Mercator::Buffer<SubProblem> outBuffer(output_size);

		BranchAndBoundKnapsack app;
	
		inBuffer.set(input_ptr, input_size);

		// app.getParams()->globalLowerBound = globalLowerBound;
		// app.getParams()->weights = d_weights;
		// app.getParams()->profits = d_profits;
		app.getParams()->maxCapacity = MAX_CAPACITY;
		app.getParams()->maxItems = MAX_ITEMS;
		
		if (num_blocks == 0) {
			num_blocks = app.getNBlocks(); // 184
			block_bounds = (double*) calloc(num_blocks, sizeof(double));
		}
		app.getParams()->numBlocks = num_blocks;
		for (unsigned i = 0; i != num_blocks; i++) {
			block_bounds[i] = globalLowerBound;
		}
		double * d_blockLowerBounds;
		hipMalloc((void**) &d_blockLowerBounds, num_blocks * sizeof(double));
		hipMemcpy(d_blockLowerBounds, block_bounds, num_blocks * sizeof(double), hipMemcpyHostToDevice); 
		app.getParams()->blockLowerBounds = d_blockLowerBounds;

		unsigned * d_weights, * d_profits;
		hipError_t cudaStatus;
		cudaStatus = hipMalloc((void**) &d_weights, MAX_ITEMS * sizeof(unsigned));
		hipMemcpy(d_weights, weights, MAX_ITEMS * sizeof(unsigned),hipMemcpyHostToDevice);
       		if (cudaStatus != hipSuccess) {
			std::cout << "hipMalloc error" << std::endl;
			hipFree(d_weights);
			return -1;
		}

		cudaStatus = hipMalloc((void**) &d_profits, MAX_ITEMS * sizeof(unsigned));
		hipMemcpy(d_profits, profits, MAX_ITEMS * sizeof(unsigned),hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			std::cout << "hipMalloc error" << std::endl;
			hipFree((void*)d_profits);
			return -1;
		}

		app.getParams()->weights = d_weights;
		app.getParams()->profits = d_profits;

		app.setSource(inBuffer);
		app.SinkNode.setSink(outBuffer);

        	app.run();

		unsigned int outsize = outBuffer.size();
		std::cout << "got " << outsize << " outputs " << std::endl;
        	if (outsize != 0) {
			outBuffer.get(output_ptr, outsize);
			if ((unsigned) index == MAX_ITEMS / 8 - 1) {
				// leaf
				// update global lower boud;
				for (size_t a = 0; a != outsize; a++) {
					if (output_ptr[a].upperBound > globalLowerBound) {
						globalLowerBound = output_ptr[a].upperBound;
					}
				}

				/*
				hipMemcpy(block_bounds, d_blockLowerBounds, num_blocks * sizeof(double), hipMemcpyDeviceToHost);
				for (size_t a = 0; a != num_blocks; a++) {
					if (block_bounds[a] > globalLowerBound) {
						globalLowerBound = block_bounds[a];
					}
				}
				*/
				std::cout << "current GPU max: " << globalLowerBound << std::endl;

			} else {
				std::copy(output_ptr, output_ptr + outsize, std::back_inserter(repo[index + 1]));
			}
		}

		// std::cout << "currentItem  " << output_ptr[0].currentItem << std::endl;
		delete [] output_ptr;
		delete [] input_ptr;
			
		hipFree((void*)d_weights);
		hipFree((void*)d_profits);
		hipFree((void*)d_blockLowerBounds);

	}
	

	std::cout << "max profit from gpu: " << globalLowerBound << std::endl;

	if (fabs(globalLowerBound - cpuGlobalLowerBound) < 0.0001) {
		std::cout << "both versions got the same result" << std::endl;
	} else {
		std::cout << "cpu got " << cpuGlobalLowerBound << " gpu got " << globalLowerBound << std::endl;
	}

	free(block_bounds);

	/*
	hipFree((void*)d_weights);
	hipFree((void*)d_profits);
	*/
        return 0;
}

